#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from dev_offsetition of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
  // TODO: implement and use this interface if necessary  
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
  // TODO: implement and use this interface if necessary  
}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary  
}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary
   

    float *device_x;
    float *device_y;
    float *device_result;

    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //
    hipMalloc(&device_x, total_elems*sizeof(float));
    hipMalloc(&device_y, total_elems*sizeof(float));
    hipMalloc(&device_result, total_elems*sizeof(float));
    hipError_t errCode_malloc = hipPeekAtLastError();
    if (errCode_malloc != hipSuccess) {
        fprintf(stderr, "malloc WARNING: A CUDA error occured: code=%d, %s\n", errCode_malloc, hipGetErrorString(errCode_malloc));
    }

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO: Compute number of thread blocks.
    // 
    int numBlocks = (total_elems+threadsPerBlock-1)/threadsPerBlock;

    double copyH2DstartTime = CycleTimer::currentSeconds();
    //
    // TODO: copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, total_elems*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, total_elems*sizeof(float), hipMemcpyHostToDevice);

    //
    // TODO: insert time here to begin timing only the kernel
    //
    double startGPUTime = CycleTimer::currentSeconds();
    double timeH2D = startGPUTime - copyH2DstartTime;
    timeCopyH2DAvg += timeH2D;

    // run saxpy_kernel on the GPU
    saxpy_kernel<<<numBlocks, threadsPerBlock>>>(total_elems, alpha, device_x, device_y, device_result);


    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();

    double endGPUTime = CycleTimer::currentSeconds();
    double timeKernel = endGPUTime - startGPUTime;
    timeKernelAvg +=timeKernel;
    
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "myWARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    
    //
    // TODO: copy result from GPU using hipMemcpy
    //
    double copyD2HstartTime = CycleTimer::currentSeconds();
    hipMemcpy(resultarray, device_result, total_elems*sizeof(float), hipMemcpyDeviceToHost);
    

    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;
    double timeD2H = endTime - copyD2HstartTime;
    timeCopyD2HAvg += timeD2H;

    //
    // TODO free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
    //

}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
