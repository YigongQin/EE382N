#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

using namespace std;
void printCudaInfo();
extern float toBW(int bytes, float sec);

struct GlobalConstants {
  int nx;
  int ny;
  int Mt;
  int nts; 
  int ictype;
  float G;
  float R;
  float delta;
  float k;
  float c_infm;
  float Dl;
  float d0;
  float W0;
  float lT;
  float lamd; 
  float tau0;
  float c_infty; 
  float R_tilde;
  float Dl_tilde; 
  float lT_tilde; 
  float eps; 
  float alpha0; 
  float dx; 
  float dt; 
  float asp_ratio; 
  float lxd;
  float lx; 
  float lyd; 
  float eta; 
  float U0; 
  // parameters that are not in the input file
  float hi;
  float cosa;
  float sina;
  float sqrt2;
  float a_s;
  float epsilon;
  float a_12;

};

__constant__ GlobalConstants cP;

// Device codes 

// boundary condition
// only use this function to access the boundary points, 
// other functions return at the boundary

__global__ void
set_BC(float* ps, float* ph, float* U, float* dpsi, int fnx, int fny){

  // find the location of boundary:
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //shared mem
  extern __shared__ float psShared[];
  float *phShared = (float*)&psShared[fnx*fny];
  float *UShared = (float*)&phShared[fnx*fny];
  psShared[index]=ps[index];
  phShared[index]=ph[index];
  UShared[index]=U[index];
  __syncthreads();
  // z=0, lx
  if (index<fnx) {
    int b_in = index+2*fnx;
    int t_out = index+(fny-1)*fnx;
    int t_in = index+(fny-3)*fnx;

    psShared[index] = psShared[b_in];
    phShared[index] = phShared[b_in];
    UShared[index] = UShared[b_in];
    dpsi[index] = dpsi[b_in];

    psShared[t_out] = psShared[t_in];
    phShared[t_out] = phShared[t_in];
    UShared[t_out] = UShared[t_in];
    dpsi[t_out] = dpsi[t_in];
  }
  if (index<fny){
    int l_out = index*fnx;
    int l_in = index*fnx + 2;
    int r_out = index*fnx + fnx -1;
    int r_in = index*fnx + fnx -3;
 
    psShared[l_out] = psShared[l_in];
    phShared[l_out] = phShared[l_in];
    UShared[l_out] = UShared[l_in];
    dpsi[l_out] = dpsi[l_in];
 
    psShared[r_out] = psShared[r_in];
    phShared[r_out] = phShared[r_in];
    UShared[r_out] = UShared[r_in];
    dpsi[r_out] = dpsi[r_in];
  }
  ps[index]=psShared[index];
  ph[index]=phShared[index];
  U[index]=UShared[index];


}

// initialization
__global__ void
initialize(float* ps_old, float* ph_old, float* U_old, float* ps_new, float* ph_new, float* U_new
           , float* x, float* y, int fnx, int fny){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx;
  int i=C-j*fnx;
  // when initialize, you need to consider C/F layout
  // if F layout, the 1D array has peroidicity of nx    
  // all the variables should be functions of x and y
  // size (nx+2)*(ny+2), x:nx, y:ny
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
    float xc = x[i];
    float yc = y[j];
    int cent = fnx/2;
    ps_old[C] = 5.625f - sqrtf( (xc-x[cent])*(xc-x[cent]) + yc*yc )/cP.W0 ;
    //if (C<1000){printf("ps %f\n",ps_old[C]);}
    ps_new[C] = ps_old[C];
    U_old[C] = cP.U0;
    U_new[C] = cP.U0;
    ph_old[C] = tanhf(ps_old[C]/cP.sqrt2);
    ph_new[C] = tanhf(ps_new[C]/cP.sqrt2);
  //  if (C<1000){printf("phi %f\n",ph_old[C]);} 
  }
}

// anisotropy functions
__device__ float
atheta(float ux, float uz){
  
   float ux2 = cP.cosa*ux + cP.sina*uz;
         ux2 = ux2*ux2;
   float uz2 = -cP.sina*ux + cP.cosa*uz;
         uz2 = uz2*uz2;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return cP.a_s*( 1.0f + cP.epsilon*(ux2*ux2 + uz2*uz2) / MAG_sq2);}
   else {return 1.0f;}
}


__device__ float
aptheta(float ux, float uz){

   float uxr = cP.cosa*ux + cP.sina*uz;
   float ux2 = uxr*uxr;
   float uzr = -cP.sina*ux + cP.cosa*uz;
   float uz2 = uzr*uzr;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return -cP.a_12*uxr*uzr*(ux2 - uz2) / MAG_sq2;}
   else {return 0.0f;}
}

// psi equation
__global__ void
rhs_psi(float* ps, float* ph, float* U, float* ps_new, float* ph_new, \
        float* y, float* dpsi, int fnx, int fny, int nt ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx; 
  int i=C-j*fnx;
  //shared mem
  extern __shared__ float psShared[];
  float *phShared = (float*)&psShared[fnx*fny];
  float *UShared = (float*)&phShared[fnx*fny];
  psShared[C]=ps[C];
  phShared[C]=ph[C];
  UShared[C]=U[C];
  __syncthreads();

  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
       // find the indices of the 8 neighbors for center
       //if (C==1000){printf("find");}
       int R=C+1;
       int L=C-1;
       int T=C+fnx;
       int B=C-fnx;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float psipjp=( psShared[C] + psShared[R] + psShared[T] + psShared[T+1] ) * 0.25f;
        float psipjm=( psShared[C] + psShared[R] + psShared[B] + psShared[B+1] ) * 0.25f;
        float psimjp=( psShared[C] + psShared[L] + psShared[T-1] + psShared[T] ) * 0.25f;
        float psimjm=( psShared[C] + psShared[L] + psShared[B-1] + psShared[B] ) * 0.25f;

        float phipjp=( phShared[C] + phShared[R] + phShared[T] + phShared[T+1] ) * 0.25f;
        float phipjm=( phShared[C] + phShared[R] + phShared[B] + phShared[B+1] ) * 0.25f;
        float phimjp=( phShared[C] + phShared[L] + phShared[T-1] + phShared[T] ) * 0.25f;
        float phimjm=( phShared[C] + phShared[L] + phShared[B-1] + phShared[B] ) * 0.25f;
        
        // ============================
        // right edge flux
        // ============================
        float psx = psShared[R]-psShared[C];
        float psz = psipjp - psipjm;
        float phx = phShared[R]-phShared[C];
        float phz = phipjp - phipjm;

        float A  = atheta( phx,phz);
        float Ap = aptheta(phx,phz);
        float JR = A * ( A*psx - Ap*psz );
        
        // ============================
        // left edge flux
        // ============================
        psx = psShared[C]-psShared[L];
        psz = psimjp - psimjm;
        phx = phShared[C]-phShared[L];
        phz = phimjp - phimjm; 

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JL = A * ( A*psx - Ap*psz );
        
        // ============================
        // top edge flux
        // ============================
        psx = psipjp - psimjp;
        psz = psShared[T]-psShared[C];
        phx = phipjp - phimjp;
        phz = phShared[T]-phShared[C];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JT = A * ( A*psz + Ap*psx );

        // ============================
        // bottom edge flux
        // ============================
        psx = psipjm - psimjm;
        psz = psShared[C]-psShared[B];
        phx = phipjm - phimjm;
        phz = phShared[C]-phShared[B];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JB = A * ( A*psz + Ap*psx );

         /*# =============================================================
        #
        # 2. EXTRA TERM: sqrt2 * atheta**2 * phi * |grad psi|^2
        #
        # =============================================================
        # d(phi)/dx  d(psi)/dx d(phi)/dz  d(psi)/dz at nodes (i,j)*/
        float phxn = ( phShared[R] - phShared[L] ) * 0.5f;
        float phzn = ( phShared[T] - phShared[B] ) * 0.5f;
        float psxn = ( psShared[R] - psShared[L] ) * 0.5f;
        float pszn = ( psShared[T] - psShared[B] ) * 0.5f;

        float A2 = atheta(phxn,phzn);
        A2 = A2*A2;
        float gradps2 = (psxn)*(psxn) + (pszn)*(pszn);
        float extra =  -cP.sqrt2 * A2 * phShared[C] * gradps2;

        /*# =============================================================
        #
        # 3. double well (transformed): sqrt2 * phi + nonlinear terms
        #
        # =============================================================*/

        float Up = (y[j]/cP.W0 - cP.R_tilde * (nt*cP.dt) )/cP.lT_tilde;

        float rhs_psi = ((JR-JL) + (JT-JB) + extra) * cP.hi*cP.hi + \
                   cP.sqrt2*phShared[C] - cP.lamd*(1.0f-phShared[C]*phShared[C])*cP.sqrt2*(UShared[C] + Up);

        /*# =============================================================
        #
        # 4. dpsi/dt term
        #
        # =============================================================*/
        float tp = (1.0f-(1.0f-cP.k)*Up);
        float tau_psi;
        if (tp >= cP.k){tau_psi = tp*A2;}
               else {tau_psi = cP.k*A2;}
        
        dpsi[C] = rhs_psi / tau_psi; 
        
        ps_new[C] = psShared[C] +  cP.dt * dpsi[C];
        ph_new[C] = tanhf(ps_new[C]/cP.sqrt2);
        //if (C==1000){printf("%f ",ph_new[C]);}
         }
} 

// U equation
__global__ void
rhs_U(float* U, float* U_new, float* ph, float* dpsi, int fnx, int fny ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx;
  int i=C-j*fnx;
  //shared mem
  extern __shared__ float phShared[];
  float *UShared = (float*)&phShared[fnx*fny];
  phShared[C]=ph[C];
  UShared[C]=U[C];
  __syncthreads();
  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
        // find the indices of the 8 neighbors for center
        int R=C+1;
        int L=C-1;
        int T=C+fnx;
        int B=C-fnx;
        float hi = cP.hi;
        float Dl_tilde = cP.Dl_tilde;
        float k = cP.k;
        float nx,nz;
        float eps = cP.eps;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float phipjp=( phShared[C] + phShared[R] + phShared[T] + phShared[T+1] ) * 0.25f;
        float phipjm=( phShared[C] + phShared[R] + phShared[B] + phShared[B+1] ) * 0.25f;
        float phimjp=( phShared[C] + phShared[L] + phShared[T-1] + phShared[T] ) * 0.25f;
        float phimjm=( phShared[C] + phShared[L] + phShared[B-1] + phShared[B] ) * 0.25f;

        float jat    = 0.5f*(1.0f+(1.0f-k)*U[C])*(1.0f-phShared[C]*phShared[C])*dpsi[C];
        /*# ============================
        # right edge flux (i+1/2, j)
        # ============================*/
        float phx = phShared[R]-phShared[C];
        float phz = phipjp - phipjm;
        float phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_ip = 0.5f*(1.0f+(1.0f-k)*U[R])*(1.0f-phShared[R]*phShared[R])*dpsi[R];	
        float UR = hi*Dl_tilde*0.5f*(2.0f - phShared[C] - phShared[R])*(UShared[R]-UShared[C]) + 0.5f*(jat + jat_ip)*nx;
    	 
    	 
        /* ============================
        # left edge flux (i-1/2, j)
        # ============================*/
        phx = phShared[C]-phShared[L];
        phz = phimjp - phimjm;
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_im = 0.5f*(1.0f+(1.0f-k)*UShared[L])*(1.0f-phShared[L]*phShared[L])*dpsi[L];
        float UL = hi*Dl_tilde*0.5f*(2.0f - phShared[C] - phShared[L])*(UShared[C]-UShared[L]) + 0.5f*(jat + jat_im)*nx;
    	 
    	 
        /*# ============================
        # top edge flux (i, j+1/2)
        # ============================*/     
        phx = phipjp - phimjp;
        phz = phShared[T]-phShared[C];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;}    	
  
        float jat_jp = 0.5f*(1.0f+(1.0f-k)*UShared[T])*(1.0f-phShared[T]*phShared[T])*dpsi[T];      
        
        float UT = hi*Dl_tilde*0.5f*(2.0f - phShared[C] - phShared[T])*(UShared[T]-UShared[C]) + 0.5f*(jat + jat_jp)*nz;
    	 
    	 
        /*# ============================
        # bottom edge flux (i, j-1/2)
        # ============================*/  
        phx = phipjm - phimjm;
        phz = phShared[C]-phShared[B];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;} 

        float jat_jm = 0.5f*(1.0f+(1.0f-k)*UShared[B])*(1.0f-phShared[B]*phShared[B])*dpsi[B];              
        float UB = hi*Dl_tilde*0.5f*(2.0f - phShared[C] - phShared[B])*(UShared[C]-UShared[B]) + 0.5f*(jat + jat_jm)*nz;
        
        float rhs_U = ( (UR-UL) + (UT-UB) ) * hi + cP.sqrt2 * jat;
        float tau_U = (1.0f+cP.k) - (1.0f-cP.k)*phShared[C];

        U_new[C] = UShared[C] + cP.dt * ( rhs_U / tau_U );

       }
}

void setup(GlobalConstants params, int fnx, int fny, float* x, float* y, float* phi, float* psi,float* U){
  // we should have already pass all the data structure in by this time
  // move those data onto device
  printCudaInfo();
  float* x_device;// = NULL;
  float* y_device;// = NULL;

  float* psi_old;// = NULL;
  float* psi_new;// = NULL;
  float* U_old;// = NULL;
  float* U_new;// = NULL;
  float* phi_old;// = NULL;
  float* phi_new;// = NULL;
  float* dpsi;// = NULL;
  // allocate x, y, phi, psi, U related params
  int length = fnx*fny;

  hipMalloc((void **)&x_device, sizeof(float) * fnx);
  hipMalloc((void **)&y_device, sizeof(float) * fny);

  hipMalloc((void **)&phi_old,  sizeof(float) * length);
  hipMalloc((void **)&psi_old,  sizeof(float) * length);
  hipMalloc((void **)&U_old,    sizeof(float) * length);
  hipMalloc((void **)&phi_new,  sizeof(float) * length);
  hipMalloc((void **)&psi_new,  sizeof(float) * length);
  hipMalloc((void **)&U_new,    sizeof(float) * length);
  hipMalloc((void **)&dpsi,    sizeof(float) * length);

  hipMemcpy(x_device, x, sizeof(float) * fnx, hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, sizeof(float) * fny, hipMemcpyHostToDevice);
  hipMemcpy(psi_old, psi, sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(phi_old, phi, sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(U_old, U, sizeof(float) * length, hipMemcpyHostToDevice);

  // pass all the read-only params into global constant
  hipMemcpyToSymbol(HIP_SYMBOL(cP), &params, sizeof(GlobalConstants));

   int blocksize_1d = 128;
   int blocksize_2d = 128;  // seems reduce the block size makes it a little faster, but around 128 is okay.
   int num_block_2d = (fnx*fny+blocksize_2d-1)/blocksize_2d;
   int num_block_1d = (fnx+fny+blocksize_1d-1)/blocksize_1d;
   printf("block size %d, # blocks %d\n", blocksize_2d, num_block_2d); 
   initialize<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, U_new, x_device, y_device, fnx, fny);
   set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_new, dpsi, fnx, fny);
   set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_old, dpsi, fnx, fny);
   hipDeviceSynchronize();
   double startTime = CycleTimer::currentSeconds();
   for (int kt=0; kt<params.Mt/2; kt++){
   //  printf("time step %d\n",kt);
     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, y_device, dpsi, fnx, fny, 2*kt );
     //hipDeviceSynchronize();
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_old, dpsi, fnx, fny);
     //hipDeviceSynchronize();
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_old, U_new, phi_new, dpsi, fnx, fny);

     //hipDeviceSynchronize();
     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_new, phi_new, U_new, psi_old, phi_old, y_device, dpsi, fnx, fny, 2*kt+1 );
     //hipDeviceSynchronize();
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_new, dpsi, fnx, fny);
     //hipDeviceSynchronize();
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_new, U_old, phi_old, dpsi, fnx, fny);
     //hipDeviceSynchronize();
   }
   hipDeviceSynchronize();
   double endTime = CycleTimer::currentSeconds();
   printf("time for %d iterations: %f s\n", params.Mt, endTime-startTime);
   hipMemcpy(psi, psi_old, length * sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(phi, phi_old, length * sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(U, U_old, length * sizeof(float),hipMemcpyDeviceToHost);

  hipFree(x_device); hipFree(y_device);
  hipFree(psi_old); hipFree(psi_new);
  hipFree(phi_old); hipFree(phi_new);
  hipFree(U_old); hipFree(U_new);
  hipFree(dpsi);  


}

/*
void time_marching(GlobalConstants params, int fnx, int fny){

   // initialize or load

   int blocksize_1d = 256;
   int blocksize_2d = 512;
   int num_block_2d = (fnx*fny+blocksize_2d-1)/blocksize_2d;
   int num_block_1d = (fnx+fny+blocksize_1d-1)/blocksize_1d;

   initialize<<< num_block_2d, blocksize_2d >>>(ps_old, ph_old, U_old, ps_new, ph_new, U_new, x_device, y_device, fnx, fny);
   

   for (int kt=0; kt<params.Mt/2; kt++){

     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, y_device, dpsi, fnx, fny, 2*kt ); 
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_old, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_old, U_new, phi_new, dpsi);


     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_new, phi_new, U_new, psi_old, phi_old, y_device, dpsi, fnx, fny, 2*kt+1 ); 
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_new, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_new, U_old, phi_old, dpsi);


   }

   
 
}*/




void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
