#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

using namespace std;

extern float toBW(int bytes, float sec);

struct GlobalConstants {
  int nx;
  int ny;
  int Mt;
  int nts; 
  int ictype;
  float G;
  float R;
  float delta;
  float k;
  float c_infm;
  float Dl;
  float d0;
  float W0;
  float lT;
  float lamd; 
  float tau0;
  float c_infty; 
  float R_tilde;
  float Dl_tilde; 
  float lT_tilde; 
  float eps; 
  float alpha0; 
  float dx; 
  float dt; 
  float asp_ratio; 
  float lxd;
  float lx; 
  float lyd; 
  float eta; 
  float U0; 
  // parameters that are not in the input file
  float hi;
  float cosa;
  float sina;
  float sqrt2;
  float a_s;
  float epsilon;
  float a_12;

};

__constant__ GlobalConstants cP;

// Device codes 

// boundary condition
// only use this function to access the boundary points, 
// other functions return at the boundary

__global__ void
set_BC(float* ps, float* ph, float* U, float* dpsi, int fnx, int fny){

  // find the location of boundary:
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // z=0, lx
  if (index<fnx) {
    int b_in = index+2*fnx;
    int t_out = index+(fny-1)*fnx;
    int t_in = index+(fny-3)*fnx;

    ps[index] = ps[b_in];
    ph[index] = ph[b_in];
    U[index] = U[b_in];
    dpsi[index] = dpsi[b_in];

    ps[t_out] = ps[t_in];
    ph[t_out] = ph[t_in];
    U[t_out] = U[t_in];
    dpsi[t_out] = dpsi[t_in];
  }
  if (index<fny){
    int l_out = index*fnx;
    int l_in = index*fnx + 2;
    int r_out = index*fnx + fnx -1;
    int r_in = index*fnx + fnx -3;
 
    ps[l_out] = ps[l_in];
    ph[l_out] = ph[l_in];
    U[l_out] = U[l_in];
    dpsi[l_out] = dpsi[l_in];
 
    ps[r_out] = ps[r_in];
    ph[r_out] = ph[r_in];
    U[r_out] = U[r_in];
    dpsi[r_out] = dpsi[r_in];
  }


}

// initialization
__global__ void
initialize(float* ps_old, float* ph_old, float* U_old, float* ps_new, float* ph_new, float* U_new
           , float* x, float* y, int fnx, int fny){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx;
  int i=C-j*fnx;
  // when initialize, you need to consider C/F layout
  // if F layout, the 1D array has peroidicity of nx    
  // all the variables should be functions of x and y
  // size (nx+2)*(ny+2), x:nx, y:ny
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
    float xc = x[i]; 
    float yc = y[j];
    int cent = fnx/2;
    ps_old[C] = 5.625f - sqrtf( (xc-x[cent])*(xc-x[cent]) + yc*yc )/cP.W0 ;
    ps_new[C] = ps_old[C];
    U_old[C] = cP.U0;
    U_new[C] = cP.U0;
    ph_old[C] = tanhf(ps_old[C]/cP.sqrt2);
    ph_new[C] = tanhf(ps_new[C]/cP.sqrt2); 
  }
}

// anisotropy functions
__device__ float
atheta(float ux, float uz){
  
   float ux2 = cP.cosa*ux + cP.sina*uz;
         ux2 = ux2*ux2;
   float uz2 = -cP.sina*ux + cP.cosa*uz;
         uz2 = uz2*uz2;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return cP.a_s*( 1.0f + cP.epsilon*(ux2*ux2 + uz2*uz2) / MAG_sq2);}
   else {return 1.0f;}
}


__device__ float
aptheta(float ux, float uz){

   float uxr = cP.cosa*ux + cP.sina*uz;
   float ux2 = uxr*uxr;
   float uzr = -cP.sina*ux + cP.cosa*uz;
   float uz2 = uzr*uzr;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return -cP.a_12*uxr*uzr*(ux2 - uz2) / MAG_sq2;}
   else {return 0.0f;}
}

// psi equation
__global__ void
rhs_psi(float* ps, float* ph, float* U, float* ps_new, float* ph_new, \
        float* y, float* dpsi, int fnx, int fny, int nt ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx; 
  int i=C-j*fnx;
  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
       // find the indices of the 8 neighbors for center
       int R=C+1;
       int L=C-1;
       int T=C+fnx;
       int B=C-fnx;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float psipjp=( ps[C] + ps[R] + ps[T] + ps[T+1] ) * 0.25f;
        float psipjm=( ps[C] + ps[R] + ps[B] + ps[B+1] ) * 0.25f;
        float psimjp=( ps[C] + ps[L] + ps[T-1] + ps[T] ) * 0.25f;
        float psimjm=( ps[C] + ps[L] + ps[B-1] + ps[B] ) * 0.25f;

        float phipjp=( ph[C] + ph[R] + ph[T] + ph[T+1] ) * 0.25f;
        float phipjm=( ph[C] + ph[R] + ph[B] + ph[B+1] ) * 0.25f;
        float phimjp=( ph[C] + ph[L] + ph[T-1] + ph[T] ) * 0.25f;
        float phimjm=( ph[C] + ph[L] + ph[B-1] + ph[B] ) * 0.25f;
        
        // ============================
        // right edge flux
        // ============================
        float psx = ps[R]-ps[C];
        float psz = psipjp - psipjm;
        float phx = ph[R]-ph[C];
        float phz = phipjp - phipjm;

        float A  = atheta( phx,phz);
        float Ap = aptheta(phx,phz);
        float JR = A * ( A*psx - Ap*psz );
        
        // ============================
        // left edge flux
        // ============================
        psx = ps[C]-ps[L];
        psz = psimjp - psimjm;
        phx = ph[C]-ph[L];
        phz = phimjp - phimjm; 

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JL = A * ( A*psx - Ap*psz );
        
        // ============================
        // top edge flux
        // ============================
        psx = psipjp - psimjp;
        psz = ps[T]-ps[C];
        phx = phipjp - phimjp;
        phz = ph[T]-ph[C];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JT = A * ( A*psz + Ap*psx );

        // ============================
        // bottom edge flux
        // ============================
        psx = psipjm - psimjm;
        psz = ps[C]-ps[B];
        phx = phipjm - phimjm;
        phz = ph[C]-ph[B];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JB = A * ( A*psz + Ap*psx );

         /*# =============================================================
        #
        # 2. EXTRA TERM: sqrt2 * atheta**2 * phi * |grad psi|^2
        #
        # =============================================================
        # d(phi)/dx  d(psi)/dx d(phi)/dz  d(psi)/dz at nodes (i,j)*/
        float phxn = ( ph[R] - ph[L] ) * 0.5f;
        float phzn = ( ph[T] - ph[B] ) * 0.5f;
        float psxn = ( ps[R] - ps[L] ) * 0.5f;
        float pszn = ( ps[T] - ps[B] ) * 0.5f;

        float A2 = atheta(phxn,phzn);
        A2 = A2*A2;
        float gradps2 = (psxn)*(psxn) + (pszn)*(pszn);
        float extra =  -cP.sqrt2 * A2 * ph[C] * gradps2;

        /*# =============================================================
        #
        # 3. double well (transformed): sqrt2 * phi + nonlinear terms
        #
        # =============================================================*/

        float Up = (y[j]/cP.W0 - cP.R_tilde * (nt*cP.dt) )/cP.lT_tilde;

        float rhs_psi = ((JR-JL) + (JT-JB) + extra) * cP.hi*cP.hi + \
                   cP.sqrt2*ph[C] - cP.lamd*(1.0f-ph[C]*ph[C])*cP.sqrt2*(U[C] + Up);

        /*# =============================================================
        #
        # 4. dpsi/dt term
        #
        # =============================================================*/
        float tp = (1.0f-(1.0f-cP.k)*Up);
        float tau_psi;
        if (tp >= cP.k){tau_psi = tp*A2;}
               else {tau_psi = cP.k*A2;}
        
        dpsi[C] = rhs_psi / tau_psi; 
        
        ps_new[C] = ps[C] +  cP.dt * dpsi[C];
        ph_new[C] = tanhf(ps_new[C]/sqrt2);
        }
} 

// U equation
__global__ void
rhs_U(float* U, float* U_new, float* ph, float* dpsi, int fnx, int fny ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx;
  int i=C-j*fnx;
  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
        // find the indices of the 8 neighbors for center
        int R=C+1;
        int L=C-1;
        int T=C+fnx;
        int B=C-fnx;
        float hi = cP.hi;
        float Dl_tilde = cP.Dl_tilde;
        float k = cP.k;
        float nx,nz;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float phipjp=( ph[C] + ph[R] + ph[T] + ph[T+1] ) * 0.25f;
        float phipjm=( ph[C] + ph[R] + ph[B] + ph[B+1] ) * 0.25f;
        float phimjp=( ph[C] + ph[L] + ph[T-1] + ph[T] ) * 0.25f;
        float phimjm=( ph[C] + ph[L] + ph[B-1] + ph[B] ) * 0.25f;

        float jat    = 0.5f*(1.0f+(1.0f-k)*U[C])*(1.0f-ph[C]*ph[C])*dpsi[C];
        /*# ============================
        # right edge flux (i+1/2, j)
        # ============================*/
        float phx = ph[R]-ph[C];
        float phz = phipjp - phipjm;
        float phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_ip = 0.5f*(1.0f+(1.0f-k)*U[R])*(1.0f-ph[R]*ph[R])*dpsi[R];	
        float UR = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[R])*(U[R]-U[C]) + 0.5f*(jat + jat_ip)*nx;
    	 
    	 
        /* ============================
        # left edge flux (i-1/2, j)
        # ============================*/
        phx = ph[C]-ph[L];
        phz = phimjp - phimjm;
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_im = 0.5f*(1.0f+(1.0f-k)*U[L])*(1.0f-ph[L]*ph[L])*dpsi[L];
        float UL = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[L])*(U[C]-U[L]) + 0.5f*(jat + jat_im)*nx;
    	 
    	 
        /*# ============================
        # top edge flux (i, j+1/2)
        # ============================*/     
        phx = phipjp - phimjp;
        phz = ph[T]-ph[C];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;}    	
  
        float jat_jp = 0.5f*(1.0f+(1.0f-k)*U[T])*(1.0f-ph[T]*ph[T])*dpsi[T];      
        
        float UT = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[T])*(U[T]-U[C]) + 0.5f*(jat + jat_jp)*nz;
    	 
    	 
        /*# ============================
        # bottom edge flux (i, j-1/2)
        # ============================*/  
        phx = phipjm - phimjm;
        phz = ph[C]-ph[B];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;} 

        float jat_jm = 0.5f*(1.0f+(1.0f-k)*U[B])*(1.0f-ph[B]*ph[B])*dpsi[B];              
        float UB = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[B])*(U[C]-U[B]) + 0.5f*(jat + jat_jm)*nz;
        
        float rhs_U = ( (UR-UL) + (UT-UB) ) * hi + cP.sqrt2 * jat;
        float tau_U = (1.0f+cP.k) - (1.0f-cP.k)*ph[C];

        U_new[C] = U[C] + cP.dt * ( rhs_U / tau_U );

       }
}

void setup(GlobalConstants params, int fnx, int fny, float* x, float* y, float* phi, float* psi,float* U){
  // we should have already pass all the data structure in by this time
  // move those data onto device

  float* x_device = NULL;
  float* y_device = NULL;

  float* psi_old = NULL;
  float* psi_new = NULL;
  float* U_old = NULL;
  float* U_new = NULL;
  float* phi_old = NULL;
  float* phi_new = NULL;
  float* dpsi = NULL;
  // allocate x, y, phi, psi, U related params
  int length = fnx*fny;

  hipMalloc(&x_device, sizeof(float) * fnx);
  hipMalloc(&y_device, sizeof(float) * fny);

  hipMalloc(&phi_old,  sizeof(float) * length);
  hipMalloc(&psi_old,  sizeof(float) * length);
  hipMalloc(&U_old,    sizeof(float) * length);
  hipMalloc(&phi_new,  sizeof(float) * length);
  hipMalloc(&psi_new,  sizeof(float) * length);
  hipMalloc(&U_new,    sizeof(float) * length);
  hipMalloc(&dpsi,    sizeof(float) * length);

  hipMemcpy(x_device, x, sizeof(float) * fnx, hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, sizeof(float) * fny, hipMemcpyHostToDevice);
  //hipMemcpy(psi_old, psi, sizeof(float) * length, hipMemcpyHostToDevice);
  //hipMemcpy(phi_old, phi, sizeof(float) * length, hipMemcpyHostToDevice);
  //hipMemcpy(U_old, U, sizeof(float) * length, hipMemcpyHostToDevice);

  // pass all the read-only params into global constant
  hipMemcpyToSymbol(HIP_SYMBOL(cP), &params, sizeof(GlobalConstants));

   int blocksize_1d = 256;
   int blocksize_2d = 512;
   int num_block_2d = (fnx*fny+blocksize_2d-1)/blocksize_2d;
   int num_block_1d = (fnx+fny+blocksize_1d-1)/blocksize_1d;

   initialize<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, U_new, x_device, y_device, fnx, fny);


   for (int kt=0; kt<params.Mt/2; kt++){

     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, y_device, dpsi, fnx, fny, 2*kt );
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_old, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_old, U_new, phi_new, dpsi, fnx, fny);


     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_new, phi_new, U_new, psi_old, phi_old, y_device, dpsi, fnx, fny, 2*kt+1 );
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_new, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_new, U_old, phi_old, dpsi, fnx, fny);

   }

  hipFree(x_device); hipFree(y_device);
  hipFree(psi_old); hipFree(psi_new);
  hipFree(phi_old); hipFree(phi_new);
  hipFree(U_old); hipFree(U_new);
  hipFree(dpsi);  


}

/*
void time_marching(GlobalConstants params, int fnx, int fny){

   // initialize or load

   int blocksize_1d = 256;
   int blocksize_2d = 512;
   int num_block_2d = (fnx*fny+blocksize_2d-1)/blocksize_2d;
   int num_block_1d = (fnx+fny+blocksize_1d-1)/blocksize_1d;

   initialize<<< num_block_2d, blocksize_2d >>>(ps_old, ph_old, U_old, ps_new, ph_new, U_new, x_device, y_device, fnx, fny);
   

   for (int kt=0; kt<params.Mt/2; kt++){

     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, y_device, dpsi, fnx, fny, 2*kt ); 
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_old, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_old, U_new, phi_new, dpsi);


     rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_new, phi_new, U_new, psi_old, phi_old, y_device, dpsi, fnx, fny, 2*kt+1 ); 
     set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_new, dpsi, fnx, fny);
     rhs_U<<< num_block_2d, blocksize_2d >>>(U_new, U_old, phi_old, dpsi);


   }

   
 
}*/




void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
